#include "hip/hip_runtime.h"
// Copyright (c) 2022 CINN Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

namespace cinn {
namespace runtime {
namespace cuda {
namespace custom {

namespace {
template <int BlockDimX, int BlockDimY, int GridDimX, typename T>
__global__ void LookupTableKernel(T* output,
                                  const T* table,
                                  const int64_t* ids,
                                  const int64_t N,
                                  const int64_t K,
                                  const int64_t D,
                                  const int64_t padding_idx) {
  constexpr int64_t kNoPadding = -1;
  int idx                      = threadIdx.x;
  int idy                      = blockIdx.x + threadIdx.y * GridDimX;

  while (idy < K) {
    int64_t id   = ids[idy];
    T* out       = output + idy * D;
    const T* tab = table + id * D;
    for (int i = idx; i < D; i += BlockDimX) {
      if (padding_flag != kNoPadding && id == padding_idx) {
        out[i] = 0;
      } else {
        out[i] = tab[i];
      }
    }
    idy += BlockDimY * GridDimX;
  }
}
}  // namespace

template <typename T>
void lookup_table(T* output,
                  const T* table,
                  const int64_t* ids,
                  int64_t row_number,
                  int64_t row_width,
                  int64_t ids_numel,
                  int64_t padding_idx,
                  hipStream_t stream) {
  dim3 threads(128, 8);
  dim3 grids(8, 1);
  LookupTableKernel<128, 8, 8>
      <<<grids, threads, 0, stream>>>(output, table, ids, row_number, row_width, ids_numel, padding_idx);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    LOG(INFO) << hipGetErrorString(error);
  }
}

template void lookup_table<float>(float* output,
                                  const float* table,
                                  const int64_t* ids,
                                  int64_t row_number,
                                  int64_t row_width,
                                  int64_t ids_numel,
                                  int64_t padding_idx,
                                  hipStream_t stream);

}  // namespace custom
}  // namespace cuda
}  // namespace runtime
}  // namespace cinn
